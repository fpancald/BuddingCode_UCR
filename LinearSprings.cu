
#include "System.h"
#include "SystemStructures.h"
#include "LinearSprings.h"

void ComputeLinearSprings(
    GeneralParams& generalParams,
    CoordInfoVecs& coordInfoVecs,
    LinearSpringInfoVecs& linearSpringInfoVecs,
    LJInfoVecs& ljInfoVecs) {
        std::cout<<"ERROR 30"<<std::endl;
        thrust::fill(linearSpringInfoVecs.tempNodeForceXReduced.begin(),linearSpringInfoVecs.tempNodeForceXReduced.end(),0.0);
        thrust::fill(linearSpringInfoVecs.tempNodeForceYReduced.begin(),linearSpringInfoVecs.tempNodeForceYReduced.end(),0.0);
        thrust::fill(linearSpringInfoVecs.tempNodeForceZReduced.begin(),linearSpringInfoVecs.tempNodeForceZReduced.end(),0.0);
        thrust::fill(linearSpringInfoVecs.tempNodeForceXUnreduced.begin(),linearSpringInfoVecs.tempNodeForceXUnreduced.end(),0.0);
        thrust::fill(linearSpringInfoVecs.tempNodeForceYUnreduced.begin(),linearSpringInfoVecs.tempNodeForceYUnreduced.end(),0.0);
        thrust::fill(linearSpringInfoVecs.tempNodeForceZUnreduced.begin(),linearSpringInfoVecs.tempNodeForceZUnreduced.end(),0.0);
    
    
        thrust::counting_iterator<int> edgeIdBegin(0);
        thrust::counting_iterator<int> edgeIdEnd(generalParams.num_of_edges);
        std::cout<<"ERROR 31"<<std::endl;

    //std::cout<<"pre linear spring: " <<coordInfoVecs.nodeForceX.size()<<std::endl;
   /* int id =ljInfoVecs.node_id_close[0];
	std::cout<<"partPos: " << coordInfoVecs.nodeLocX[id]<< " "<< coordInfoVecs.nodeLocY[id] << " "<< coordInfoVecs.nodeLocZ[id] << std::endl;
	std::cout<<"partForce: " << coordInfoVecs.nodeForceX[id]<< " "<< coordInfoVecs.nodeForceY[id] << " "<< coordInfoVecs.nodeForceZ[id] << std::endl;
      */  

    linearSpringInfoVecs.linear_spring_energy=
    thrust::transform_reduce(  
        thrust::make_zip_iterator(
            thrust::make_tuple(
                edgeIdBegin,
                coordInfoVecs.edges2Nodes_1.begin(),
                coordInfoVecs.edges2Nodes_2.begin(),
                linearSpringInfoVecs.edge_initial_length.begin())),
        thrust::make_zip_iterator( 
            thrust::make_tuple(
                edgeIdBegin,
                coordInfoVecs.edges2Nodes_1.begin(),
                coordInfoVecs.edges2Nodes_2.begin(), 
                linearSpringInfoVecs.edge_initial_length.begin())) + generalParams.num_of_edges,
        LinearSpringFunctor(
            linearSpringInfoVecs.spring_constant, 
            linearSpringInfoVecs.spring_constant_weak,
            thrust::raw_pointer_cast(generalParams.edges_in_upperhem.data()),
            thrust::raw_pointer_cast(coordInfoVecs.nodeLocX.data()),
            thrust::raw_pointer_cast(coordInfoVecs.nodeLocY.data()),
            thrust::raw_pointer_cast(coordInfoVecs.nodeLocZ.data()),

            thrust::raw_pointer_cast(linearSpringInfoVecs.tempNodeIdUnreduced.data()),
            thrust::raw_pointer_cast(linearSpringInfoVecs.tempNodeForceXUnreduced.data()),
            thrust::raw_pointer_cast(linearSpringInfoVecs.tempNodeForceYUnreduced.data()),
            thrust::raw_pointer_cast(linearSpringInfoVecs.tempNodeForceZUnreduced.data()) ),
        0.0, thrust::plus<double>() ); 
        std::cout<<"ERROR 32"<<std::endl;
        //for (int i = 0; i < linearSpringInfoVecs.tempNodeIdUnreduced.size(); i++){
          //  std::cout<<"tempNodeIdUnreduced"<<linearSpringInfoVecs.tempNodeIdUnreduced[i]<<std::endl;
        //}
      //std::cout<<"linear energy from spring.cu: "<< linearSpringInfoVecs.linear_spring_energy<<std::endl;
    //now we have un reduced forces. Sort by id and reduce. 
    //key, then value. Each vector returns sorted		
    thrust::sort_by_key(linearSpringInfoVecs.tempNodeIdUnreduced.begin(), linearSpringInfoVecs.tempNodeIdUnreduced.begin() + (generalParams.num_of_edges*linearSpringInfoVecs.factor),
        thrust::make_zip_iterator(
            thrust::make_tuple(
                linearSpringInfoVecs.tempNodeForceXUnreduced.begin(),
                linearSpringInfoVecs.tempNodeForceYUnreduced.begin(),
                linearSpringInfoVecs.tempNodeForceZUnreduced.begin())), thrust::less<int>());
                std::cout<<"ERROR 33"<<std::endl;
   /* std::cout<<"mid1 linear spring: " <<coordInfoVecs.nodeForceX.size()<<std::endl;
    std::cout<<"partPos: " << coordInfoVecs.nodeLocX[id]<< " "<< coordInfoVecs.nodeLocY[id] << " "<< coordInfoVecs.nodeLocZ[id] << std::endl;
	std::cout<<"partForce: " << coordInfoVecs.nodeForceX[id]<< " "<< coordInfoVecs.nodeForceY[id] << " "<< coordInfoVecs.nodeForceZ[id] << std::endl;
	*/
    int endKey = thrust::get<0>(
        thrust::reduce_by_key(
            linearSpringInfoVecs.tempNodeIdUnreduced.begin(), 
            linearSpringInfoVecs.tempNodeIdUnreduced.end(),
        thrust::make_zip_iterator(
            thrust::make_tuple(
                linearSpringInfoVecs.tempNodeForceXUnreduced.begin(),
                linearSpringInfoVecs.tempNodeForceYUnreduced.begin(),
                linearSpringInfoVecs.tempNodeForceZUnreduced.begin())),
            linearSpringInfoVecs.tempNodeIdReduced.begin(),
        thrust::make_zip_iterator(
            thrust::make_tuple(
                linearSpringInfoVecs.tempNodeForceXReduced.begin(),
                linearSpringInfoVecs.tempNodeForceYReduced.begin(),
                linearSpringInfoVecs.tempNodeForceZReduced.begin())),
        thrust::equal_to<int>(), CVec3Add())) - linearSpringInfoVecs.tempNodeIdReduced.begin();//binary_pred, binary_op 
        std::cout<<"ERROR 34"<<std::endl;
/*
    std::cout<<"mid2 linear spring: " <<coordInfoVecs.nodeForceX.size()<<std::endl;
    std::cout<<"partPos: " << coordInfoVecs.nodeLocX[id]<< " "<< coordInfoVecs.nodeLocY[id] << " "<< coordInfoVecs.nodeLocZ[id] << std::endl;
	std::cout<<"partForce: " << coordInfoVecs.nodeForceX[id]<< " "<< coordInfoVecs.nodeForceY[id] << " "<< coordInfoVecs.nodeForceZ[id] << std::endl;
	*/
    //apply reduced force to all nodes. 
    thrust::for_each(
        thrust::make_zip_iterator(//1st begin
            thrust::make_tuple(
                linearSpringInfoVecs.tempNodeIdReduced.begin(),
                linearSpringInfoVecs.tempNodeForceXReduced.begin(),
                linearSpringInfoVecs.tempNodeForceYReduced.begin(),
                linearSpringInfoVecs.tempNodeForceZReduced.begin())),
        thrust::make_zip_iterator(//1st end
            thrust::make_tuple(
                linearSpringInfoVecs.tempNodeIdReduced.begin(),
                linearSpringInfoVecs.tempNodeForceXReduced.begin(),
                linearSpringInfoVecs.tempNodeForceYReduced.begin(),
                linearSpringInfoVecs.tempNodeForceZReduced.begin())) + endKey,
        AddForceFunctor (
            generalParams.num_of_nodes,
            thrust::raw_pointer_cast(coordInfoVecs.nodeForceX.data()),
            thrust::raw_pointer_cast(coordInfoVecs.nodeForceY.data()),
            thrust::raw_pointer_cast(coordInfoVecs.nodeForceZ.data())));
    /*
    std::cout<<"post linear spring: " <<coordInfoVecs.nodeForceX.size()<<std::endl;
    std::cout<<"partPos: " << coordInfoVecs.nodeLocX[id]<< " "<< coordInfoVecs.nodeLocY[id] << " "<< coordInfoVecs.nodeLocZ[id] << std::endl;
	std::cout<<"partForce: " << coordInfoVecs.nodeForceX[id]<< " "<< coordInfoVecs.nodeForceY[id] << " "<< coordInfoVecs.nodeForceZ[id] << std::endl;
	*/
    std::cout<<"ERROR 35"<<std::endl;
};